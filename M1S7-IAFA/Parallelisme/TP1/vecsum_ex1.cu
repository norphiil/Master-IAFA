
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void vecADDKernel(unsigned int *d_vec, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int offset = n / 2; offset > 1; offset = offset / 2)
    {
        if (index < offset)
        {
            d_vec[index] += d_vec[index + offset];
        }
        __syncthreads();
    }
}

void vecADD(unsigned int *vec, int n)
{
    int bsize = 1024;
    int gsize = ((n + bsize - 1) / bsize);
    int vecSize = n * sizeof(unsigned int);
    unsigned int *d_vec;
    hipMalloc((void **)&d_vec, vecSize);
    hipMemcpy(d_vec, vec, vecSize, hipMemcpyHostToDevice);
    vecADDKernel<<<gsize, bsize>>>(d_vec, n);
    hipMemcpy(vec, d_vec, vecSize, hipMemcpyDeviceToHost);
    hipFree(d_vec);
}

int main(int argc, char **argv)
{
    int n = 1024;
    FILE *ptr;
    unsigned int vec[n];
    ptr = fopen("result.txt", "r");

    if (NULL == ptr)
    {
        printf("file can't be opened \n");
    }

    int i = 0;
    while (!feof(ptr))
    {
        vec[i] = (int)fgetc(ptr);
        i++;
    }
    fclose(ptr);

    vecADD(vec, n);

    printf("%i ", vec[0]);
}